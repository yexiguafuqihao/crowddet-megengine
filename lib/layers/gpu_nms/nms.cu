#include "hip/hip_runtime.h"
#include "megbrain_pubapi.h"
#include <iostream>
#include <vector>
#include <assert.h>

#define DIVUP(m,n) ((m) / (n) + ((m) % (n) > 0))
#define CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    if (error != hipSuccess) { \
      std::cout << " " << hipGetErrorString(error); \
    } \
  } while (0)
#define CUDA_POST_KERNEL_CHECK CUDA_CHECK(hipPeekAtLastError())

int const threadsPerBlock = sizeof(unsigned long long) * 8; // 64

__device__ inline float devIoU(float const * const a, float const * const b) {
  float left = max(a[0], b[0]), right = min(a[2], b[2]);
  float top = max(a[1], b[1]), bottom = min(a[3], b[3]);
  float width = max(right - left + 1, 0.f), height = max(bottom - top + 1, 0.f);
  float interS = width * height;
  float Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
  float Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
  return interS / (Sa + Sb - interS);
}

__global__ void nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           const float *dev_boxes, unsigned long long *dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  if (row_start > col_start) return;

  const int row_size =
        min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ float block_boxes[threadsPerBlock * 5];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 5 + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 0];
    block_boxes[threadIdx.x * 5 + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 1];
    block_boxes[threadIdx.x * 5 + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 2];
    block_boxes[threadIdx.x * 5 + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 3];
    block_boxes[threadIdx.x * 5 + 4] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 4];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const float *cur_box = dev_boxes + cur_box_idx * 5;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (devIoU(cur_box, block_boxes + i * 5) > nms_overlap_thresh) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = DIVUP(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

template <int unroll = 4>
static inline void cpu_unroll_for(unsigned long long *dst, const unsigned long long *src, int n) {
    int nr_out = (n - n % unroll) / unroll;
    for (int i = 0; i < nr_out; ++i) {
#pragma unroll
        for (int j = 0; j < unroll; ++j) {
            *(dst++) |= *(src++);
        }
    }
    for (int j = 0; j < n % unroll; ++j) {
        *(dst++) |= *(src++);
    }
}

using std::vector;
// const int nr_init_box = 8000;
// vector<unsigned long long> _mask_host(nr_init_box * (nr_init_box / threadsPerBlock));
// vector<unsigned long long> _remv(nr_init_box / threadsPerBlock);
// vector<int> _keep_out(nr_init_box);

// NOTE: If we directly use this lib in nmp.py, we will meet the same _mask_host and other 
// objects, which is not safe for multi-processing programs.

class HostDevice{
protected:
    static const int nr_init_box = 8000;
public:
    vector<unsigned long long> mask_host;
    vector<unsigned long long> remv;
    vector<int> keep_out;

    HostDevice(): mask_host(nr_init_box * (nr_init_box / threadsPerBlock)), remv(nr_init_box / threadsPerBlock), keep_out(nr_init_box){}
};

extern "C"{
    using MGBDevTensor = mgb::pubapi::DeviceTensor;
    using std::cout;

    void * CreateHostDevice(){
        return new HostDevice();
    }
   
    int NMSForwardGpu(void* box_ptr, void* mask_ptr, void* output_ptr, void* output_num_ptr, float iou_threshold, int max_output, void* host_device_ptr){
        auto box_tensor = mgb::pubapi::as_versioned_obj<MGBDevTensor>(box_ptr);
        auto mask_tensor= mgb::pubapi::as_versioned_obj<MGBDevTensor>(mask_ptr);
        auto output_tensor = mgb::pubapi::as_versioned_obj<MGBDevTensor>(output_ptr);
        auto output_num_tensor = mgb::pubapi::as_versioned_obj<MGBDevTensor>(output_num_ptr);

        // auto hip_stream = static_cast<hipStream_t> (box_tensor->desc.cuda_ctx.stream);
         auto hip_stream = static_cast<hipStream_t> (output_tensor->desc.cuda_ctx.stream);
        // assert(box_tensor->desc.shape[0] == output_tensor->desc.shape[0]);

        // cout << "box_tensor.ndim: " << box_tensor->desc.ndim << "\n";
        // cout << "box_tensor.shape_0: " << box_tensor->desc.shape[0] << "\n";
        // cout << "box_tensor.shape_1: " << box_tensor->desc.shape[1] << "\n";
        int box_num = box_tensor->desc.shape[0];
        int box_dim = box_tensor->desc.shape[1];
        assert(box_dim == 5);

        const int col_blocks = DIVUP(box_num, threadsPerBlock);
        // cout << "mask_dev size: " << box_num * col_blocks * sizeof(unsigned long long) << "\n";
        // cout << "mask_ptr size: " << mask_tensor->desc.shape[0] * sizeof(int) << "\n";
        // cout << "mask shape : " << mask_tensor->desc.shape[0] << "\n";

        dim3 blocks(DIVUP(box_num, threadsPerBlock), DIVUP(box_num, threadsPerBlock));
        // dim3 blocks(col_blocks, col_blocks);
        dim3 threads(threadsPerBlock);
        // cout << "sizeof unsigned long long " << sizeof(unsigned long long) << "\n"; 
        float* dev_box = static_cast<float*> (box_tensor->desc.dev_ptr);
        unsigned long long* dev_mask = static_cast<unsigned long long*> (mask_tensor->desc.dev_ptr);
        int * dev_output = static_cast<int*> (output_tensor->desc.dev_ptr);

        CUDA_CHECK(hipMemsetAsync(dev_mask, 0, mask_tensor->desc.shape[0] * sizeof(int), hip_stream));
        // CUDA_CHECK(hipMemsetAsync(dev_output, 0, output_tensor->desc.shape[0] * sizeof(int), hip_stream));
        nms_kernel<<<blocks, threads, 0, hip_stream>>>(box_num, iou_threshold, dev_box, dev_mask);
        // hipDeviceSynchronize();
        
        // get the host device vectors
        HostDevice* host_device =  static_cast<HostDevice* >(host_device_ptr);
        vector<unsigned long long>& _mask_host = host_device->mask_host;
        vector<unsigned long long>& _remv      = host_device->remv;
        vector<int>& _keep_out                 = host_device->keep_out;


        int current_mask_host_size = box_num * col_blocks;
        if(_mask_host.capacity() < current_mask_host_size){
            _mask_host.reserve(current_mask_host_size);
        }
        CUDA_CHECK(hipMemcpyAsync(&_mask_host[0], dev_mask, sizeof(unsigned long long) * box_num * col_blocks, hipMemcpyDeviceToHost, hip_stream));
        // cout << "\n m_host site: " << static_cast<void *> (&_mask_host[0]) << "\n";

        if(_remv.capacity() < col_blocks){
            _remv.reserve(col_blocks);
        }
        if(_keep_out.capacity() < box_num){
            _keep_out.reserve(box_num);
        }
        if(max_output < 0){
            max_output = box_num;
        }
        memset(&_remv[0], 0, sizeof(unsigned long long) * col_blocks);
        CUDA_CHECK(hipStreamSynchronize(hip_stream));

        // do the cpu reduce
        int num_to_keep = 0;
        for (int i = 0; i < box_num; i++) {
            int nblock = i / threadsPerBlock;
            int inblock = i % threadsPerBlock;

            if (!(_remv[nblock] & (1ULL << inblock))) {
                _keep_out[num_to_keep++] = i;
                if(num_to_keep == max_output){
                    break;
                }
                // NOTE: here we need add nblock to pointer p
                unsigned long long *p = &_mask_host[0] + i * col_blocks + nblock;
                unsigned long long *q = &_remv[0] + nblock;
                cpu_unroll_for(q, p, col_blocks - nblock);
            }
        }
        CUDA_CHECK(hipMemcpyAsync(dev_output, &_keep_out[0], num_to_keep * sizeof(int), hipMemcpyHostToDevice, hip_stream));
        int* dev_output_num = static_cast<int*>(output_num_tensor->desc.dev_ptr);
        CUDA_CHECK(hipMemcpyAsync(dev_output_num, &num_to_keep, sizeof(int), hipMemcpyHostToDevice, hip_stream));
        // CUDA_CHECK(hipStreamSynchronize(hip_stream));
        return num_to_keep;
    }
}
